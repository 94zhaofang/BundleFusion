#include "hip/hip_runtime.h"

#include "CUDAImageUtil.h"

#include "mlibCuda.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)




template<class T> void CUDAImageUtil::copy(T* d_output, T* d_input, unsigned int width, unsigned int height) {
	MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(T)*width*height, hipMemcpyDeviceToDevice));
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float bilinearInterpolationFloat(float x, float y, const float* d_input, unsigned int imageWidth, unsigned int imageHeight)
{
	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta = y - p00.y;

	float s0 = 0.0f; float w0 = 0.0f;
	if (p00.x < imageWidth && p00.y < imageHeight) { float v00 = d_input[p00.y*imageWidth + p00.x]; if (v00 != MINF) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x < imageWidth && p10.y < imageHeight) { float v10 = d_input[p10.y*imageWidth + p10.x]; if (v10 != MINF) { s0 += alpha *v10; w0 += alpha; } }

	float s1 = 0.0f; float w1 = 0.0f;
	if (p01.x < imageWidth && p01.y < imageHeight) { float v01 = d_input[p01.y*imageWidth + p01.x]; if (v01 != MINF) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x < imageWidth && p11.y < imageHeight) { float v11 = d_input[p11.y*imageWidth + p11.x]; if (v11 != MINF) { s1 += alpha *v11; w1 += alpha; } }

	const float p0 = s0 / w0;
	const float p1 = s1 / w1;

	float ss = 0.0f; float ww = 0.0f;
	if (w0 > 0.0f) { ss += (1.0f - beta)*p0; ww += (1.0f - beta); }
	if (w1 > 0.0f) { ss += beta *p1; ww += beta; }

	if (ww > 0.0f) return ss / ww;
	else		  return MINF;
}

//template<class T>
//__global__ void resample_Kernel(T* d_output, T* d_input, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
//{
//	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
//
//	if (x < outputWidth && y < outputHeight)
//	{
//		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
//		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);
//
//		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
//		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);
//
//		if (xInput < inputWidth && yInput < inputHeight)
//		{
//			if (std::is_same<T, float>::value) {
//				d_output[y*outputWidth + x] = (T)bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, (float*)d_input, inputWidth, inputHeight);
//			}
//			else if (std::is_same<T, uchar4>::value) {
//				d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
//			}
//			else {
//				//static_assert(false, "bla");
//			}
//		}
//	}
//}
//
//template<class T> void CUDAImageUtil::resample(T* d_output, unsigned int outputWidth, unsigned int outputHeight, T* d_input, unsigned int inputWidth, unsigned int inputHeight) {
//
//	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
//	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);
//
//	resample_Kernel << <gridSize, blockSize >> >(d_output, d_input, inputWidth, inputHeight, outputWidth, outputHeight);
//
//#ifdef _DEBUG
//	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
//	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
//#endif
//}


__global__ void resampleFloat_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
			//d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
		}
	}
}

void CUDAImageUtil::resampleFloat(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}


__global__ void resampleUCHAR4_Kernel(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
		}
	}
}

void CUDAImageUtil::resampleUCHAR4(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__host__ __device__
float convertToIntensity(const uchar4& c) {
	return (0.299f*c.x + 0.587f*c.y + 0.114f*c.z) / 255.0f;
}

__global__ void resampleToIntensity_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = convertToIntensity(d_input[yInput*inputWidth + xInput]);
		}
	}
}

void CUDAImageUtil::resampleToIntensity(float* d_output, unsigned int outputWidth, unsigned int outputHeight, uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleToIntensity_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}