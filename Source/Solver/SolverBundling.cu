#include "hip/hip_runtime.h"
#include <iostream>

#include "SolverBundlingParameters.h"
#include "SolverBundlingState.h"
#include "SolverBundlingUtil.h"
#include "SolverBundlingEquations.h"
#include "../../SiftGPU/CUDATimer.h"

#include <conio.h>


#define THREADS_PER_BLOCK_DENSE_DEPTH_X 32
#define THREADS_PER_BLOCK_DENSE_DEPTH_Y 4 
#define THREADS_PER_BLOCK_DENSE_DEPTH_FLIP 64

/////////////////////////////////////////////////////////////////////////
// Dense Depth Term
/////////////////////////////////////////////////////////////////////////
__global__ void FlipJtJ_Kernel(unsigned int total, unsigned int dim, float* d_JtJ)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total) {
		const unsigned int x = idx % dim;
		const unsigned int y = idx / dim;
		if (x > y) {
			d_JtJ[y * dim + x] = d_JtJ[x * dim + y];
		}
	}
}
__device__ bool findDenseDepthCorr(unsigned int idx, unsigned int imageWidth, unsigned int imageHeight,
	float distThresh, float normalThresh, /*float colorThresh,*/ const float4x4& transform, const float4x4& intrinsics,
	const float4* tgtCamPos, const float4* tgtNormals, //const uchar4* tgtColor,
	const float4* srcCamPos, const float4* srcNormals, //const uchar4* srcColor,
	float depthMin, float depthMax, float4& camPosSrcToTgt, float4& camPosTgt, float4& normalTgt)
{
	const float4 cposj = srcCamPos[idx];
	//if (debugPrint) printf("cam pos j = %f %f %f\n", cposj.x, cposj.y, cposj.z);
	if (cposj.z > depthMin && cposj.z < depthMax) {
		float4 nrmj = srcNormals[idx];
		//if (debugPrint) printf("normal j = %f %f %f\n", nrmj.x, nrmj.y, nrmj.z);
		if (nrmj.x != MINF) {
			nrmj = transform * nrmj;
			camPosSrcToTgt = transform * cposj;
			float3 proj = intrinsics * make_float3(camPosSrcToTgt.x, camPosSrcToTgt.y, camPosSrcToTgt.z);
			int2 screenPos = make_int2((int)roundf(proj.x / proj.z), (int)roundf(proj.y / proj.z));
			//const uchar4& colorj = srcColor[idx];
			//if (debugPrint) {
			//	printf("cam pos j2i = %f %f %f\n", camPosSrcToTgt.x, camPosSrcToTgt.y, camPosSrcToTgt.z);
			//	printf("proj %f %f %f -> %f %f\n", proj.x, proj.y, proj.z, proj.x / proj.z, proj.y / proj.z);
			//	printf("screen pos = %d %d\n", screenPos.x, screenPos.y);
			//}
			if (screenPos.x >= 0 && screenPos.y >= 0 && screenPos.x < (int)imageWidth && screenPos.y < (int)imageHeight) {
				camPosTgt = tgtCamPos[screenPos.y * imageWidth + screenPos.x];
				//if (debugPrint) printf("cam pos i = %f %f %f\n", camPosTgt.x, camPosTgt.y, camPosTgt.z);
				if (camPosTgt.z > depthMin && camPosTgt.z < depthMax) {
					normalTgt = tgtNormals[screenPos.y * imageWidth + screenPos.x];
					//if (debugPrint) printf("normal i = %f %f %f\n", normalTgt.x, normalTgt.y, normalTgt.z);
					if (normalTgt.x != MINF) {
						float dist = length(camPosSrcToTgt - camPosTgt);
						float dNormal = dot(nrmj, normalTgt);
						//const uchar4& colori = tgtColor[idx];
						//float dColor = length(make_float3(colorj.x - colori.x, colorj.y - colori.y, colorj.z - colori.z) / 255.0f);
						//if (debugPrint) printf("dist = %f, dnormal = %f, colorDist = %f\n", dist, dNormal, dColor);
						//if (debugPrint) printf("dist = %f, dnormal = %f\n", dist, dNormal);
						if (dNormal >= normalThresh && dist <= distThresh /*&& dColor <= colorThresh*/) {
							return true;
						}
					}
				}
			} // valid projection
		} // valid src normal
	} // valid src camera position
	return false;
}
__global__ void FindDenseCorrespondences_Kernel(SolverInput input, SolverState state, SolverParameters parameters)
{
	// image indices
	unsigned int i, j; // project from j to i
	unsigned int imPairIdx;
	if (parameters.useDenseDepthAllPairwise) {
		i = blockIdx.x; j = blockIdx.y; // all pairwise
		if (i >= j) return;
		imPairIdx = i * gridDim.x + j;
	}
	else {
		i = blockIdx.x; j = i + 1; // frame-to-frame
		imPairIdx = i;
	}

	const unsigned int idx = threadIdx.y * THREADS_PER_BLOCK_DENSE_DEPTH_X + threadIdx.x;
	const unsigned int gidx = idx * gridDim.z + blockIdx.z;
	
	if (gidx < (input.denseDepthWidth * input.denseDepthHeight)) {
		float4x4 transform_i = evalRtMat(state.d_xRot[i], state.d_xTrans[i]);
		float4x4 transform_j = evalRtMat(state.d_xRot[j], state.d_xTrans[j]);
		float4x4 invTransform_i = transform_i.getInverse(); //TODO pre-compute the inverts

		float4x4 transform = invTransform_i * transform_j;

		// find correspondence
		float4 camPosSrcToTgt, camPosTgt, normalTgt;
		if (findDenseDepthCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDepthDistThresh, parameters.denseDepthNormalThresh, /*parameters.denseDepthColorThresh,*/ transform, input.depthIntrinsics,
			input.d_depthFrames[i].d_cameraposDownsampled, input.d_depthFrames[i].d_normalsDownsampled, //input.d_depthFrames[i].d_colorDownsampled,//target
			input.d_depthFrames[j].d_cameraposDownsampled, input.d_depthFrames[j].d_normalsDownsampled, //input.d_depthFrames[j].d_colorDownsampled,//source
			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrcToTgt, camPosTgt, normalTgt)) { //i tgt, j src
			atomicAdd(&state.d_denseCorrCounts[imPairIdx], 1);
		} // found correspondence
	} // valid image pixel
}

__global__ void BuildDenseDepthSystem_Kernel(SolverInput input, SolverState state, SolverParameters parameters)
{
	// image indices
	unsigned int i, j; // project from j to i
	unsigned int imPairIdx;
	if (parameters.useDenseDepthAllPairwise) {
		i = blockIdx.x; j = blockIdx.y; // all pairwise
		if (i >= j) return;
		imPairIdx = i * gridDim.x + j;
	}
	else {
		i = blockIdx.x; j = i + 1; // frame-to-frame
		imPairIdx = i;
	}

	const unsigned int idx = threadIdx.y * THREADS_PER_BLOCK_DENSE_DEPTH_X + threadIdx.x;
	const unsigned int gidx = idx * gridDim.z + blockIdx.z;
	
	if (gidx < (input.denseDepthWidth * input.denseDepthHeight)) {
		float4x4 transform_i = evalRtMat(state.d_xRot[i], state.d_xTrans[i]);
		float4x4 transform_j = evalRtMat(state.d_xRot[j], state.d_xTrans[j]);
		float4x4 invTransform_i = transform_i.getInverse(); //TODO unncessary invert for pairwise?

		float4x4 transform = invTransform_i * transform_j;

		//!!!debugging
		const unsigned int x = gidx % input.denseDepthWidth; const unsigned int y = gidx / input.denseDepthWidth;
		float4 camPosSrcToTgt = make_float4(MINF, MINF, MINF, MINF), camPosTgt = make_float4(MINF, MINF, MINF, MINF), normalTgt = make_float4(MINF, MINF, MINF, MINF);
		//!!!debugging

		// find correspondence
		//float4 camPosSrcToTgt, camPosTgt, normalTgt;
		if (findDenseDepthCorr(gidx, input.denseDepthWidth, input.denseDepthHeight,
			parameters.denseDepthDistThresh, parameters.denseDepthNormalThresh, /*parameters.denseDepthColorThresh,*/ transform, input.depthIntrinsics,
			input.d_depthFrames[i].d_cameraposDownsampled, input.d_depthFrames[i].d_normalsDownsampled, //input.d_depthFrames[i].d_colorDownsampled,//target
			input.d_depthFrames[j].d_cameraposDownsampled, input.d_depthFrames[j].d_normalsDownsampled, //input.d_depthFrames[j].d_colorDownsampled,//source
			parameters.denseDepthMin, parameters.denseDepthMax, camPosSrcToTgt, camPosTgt, normalTgt)) { //i tgt, j src
			// residual
			float4 diff = camPosTgt - camPosSrcToTgt;
			float res = dot(diff, normalTgt);

			// jacobian
			const float4& camPosSrc = input.d_depthFrames[j].d_cameraposDownsampled[gidx];
			matNxM<1, 6> jacobianBlockRow_i, jacobianBlockRow_j;
			if (i > 0) computeJacobianBlockRow_i(jacobianBlockRow_i, state.d_xRot[i], state.d_xTrans[i], transform_j, camPosSrc, normalTgt);
			if (j > 0) computeJacobianBlockRow_j(jacobianBlockRow_j, state.d_xRot[j], state.d_xTrans[j], invTransform_i, camPosSrc, normalTgt);
			float weight = max(0.0f, 0.5f*((1.0f - length(diff) / parameters.denseDepthDistThresh) + (1.0f - camPosTgt.z / parameters.denseDepthMax)));
			float imPairWeight = 1.0f / state.d_denseCorrCounts[imPairIdx];

			addToLocalSystem(state.d_depthJtJ, state.d_depthJtr, input.numberOfImages * 6,
				jacobianBlockRow_i, jacobianBlockRow_j, i, j, res, parameters.weightDenseDepth * weight * imPairWeight);

			//!!!debugging
			atomicAdd(state.d_sumResidual, parameters.weightDenseDepth * weight * imPairWeight * res * res);
			atomicAdd(state.d_corrCount, 1);
			//if (i == 0 && j == 1 && x == 47 && y == 53) {
			if (i > 0 && (isnan(jacobianBlockRow_i(0)) || isnan(jacobianBlockRow_i(1)) || isnan(jacobianBlockRow_i(2)) || isnan(jacobianBlockRow_i(3)) || isnan(jacobianBlockRow_i(4)) || isnan(jacobianBlockRow_i(5))) ||
				j > 0 && (isnan(jacobianBlockRow_j(0)) || isnan(jacobianBlockRow_j(1)) || isnan(jacobianBlockRow_j(2)) || isnan(jacobianBlockRow_j(3)) || isnan(jacobianBlockRow_j(4)) || isnan(jacobianBlockRow_j(5))) ||
				isnan(res) || isnan(weight)) {
				printf("ERROR NaN (%d,%d,%d,%d) %f %f | (%f %f %f %f %f %f) (%f %f %f %f %f %f)\n", i, j, x, y, res, weight,
					jacobianBlockRow_i(0), jacobianBlockRow_i(1), jacobianBlockRow_i(2),
					jacobianBlockRow_i(3), jacobianBlockRow_i(4), jacobianBlockRow_i(5),
					jacobianBlockRow_j(0), jacobianBlockRow_j(1), jacobianBlockRow_j(2),
					jacobianBlockRow_j(3), jacobianBlockRow_j(4), jacobianBlockRow_j(5));
			//	printf("-----------\n");
			//	printf("images (%d, %d) at (%d, %d)\n", i, j, x, y);
			//	//printf("transform i:\n"); transform_i.print();
			//	//printf("inv transform i:\n"); invTransform_i.print();
			//	//printf("transform j:\n"); transform_j.print();
			//	//printf("transform:\n"); transform.print();
			//	printf("cam pos src: %f %f %f\n", camPosSrc.x, camPosSrc.y, camPosSrc.z);
			//	printf("cam pos src to tgt: %f %f %f\n", camPosSrcToTgt.x, camPosSrcToTgt.y, camPosSrcToTgt.z);
			//	printf("cam pos tgt: %f %f %f\n", camPosTgt.x, camPosTgt.y, camPosTgt.z);
			//	printf("normal tgt: %f %f %f\n", normalTgt.x, normalTgt.y, normalTgt.z);
			//	printf("diff = %f %f %f %f\n", diff.x, diff.y, diff.z, diff.w);
			//	printf("res = %f\n", res);
			//	printf("weight = %f\n", parameters.weightDenseDepth * weight);
			//	printf("jac i %f %f %f %f %f %f\n", jacobianBlockRow_i(0), jacobianBlockRow_i(1), jacobianBlockRow_i(2),
			//		jacobianBlockRow_i(3), jacobianBlockRow_i(4), jacobianBlockRow_i(5));
			//	printf("jac j %f %f %f %f %f %f\n", jacobianBlockRow_j(0), jacobianBlockRow_j(1), jacobianBlockRow_j(2),
			//		jacobianBlockRow_j(3), jacobianBlockRow_j(4), jacobianBlockRow_j(5));
			}
			//!!!debugging
		} // found correspondence
	} // valid image pixel
}

void BuildDenseDepthSystem(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	const unsigned int N = input.numberOfImages;

	const int threadsPerBlock = THREADS_PER_BLOCK_DENSE_DEPTH_X * THREADS_PER_BLOCK_DENSE_DEPTH_Y;
	const int reductionGlobal = (input.denseDepthWidth*input.denseDepthHeight + threadsPerBlock - 1) / threadsPerBlock;
	const int sizeJtr = 6 * N;
	const int sizeJtJ = sizeJtr * sizeJtr;

	dim3 grid;
	if (parameters.useDenseDepthAllPairwise) grid = dim3(N, N, reductionGlobal); // pairwise
	else grid = dim3(N - 1, 1, reductionGlobal); // for frame-to-frame
	dim3 block(THREADS_PER_BLOCK_DENSE_DEPTH_X, THREADS_PER_BLOCK_DENSE_DEPTH_Y);

	if (timer) timer->startEvent("BuildDenseDepthSystem");

	//!!!debugging
	cutilSafeCall(hipMemset(state.d_corrCount, 0, sizeof(int)));
	cutilSafeCall(hipMemset(state.d_sumResidual, 0, sizeof(float)));
	//!!!debugging
	
	cutilSafeCall(hipMemset(state.d_denseCorrCounts, 0, sizeof(int) * input.maxNumDenseImPairs));
	cutilSafeCall(hipMemset(state.d_depthJtJ, 0, sizeof(float) * sizeJtJ)); //TODO check if necessary
	cutilSafeCall(hipMemset(state.d_depthJtr, 0, sizeof(float) * sizeJtr));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif		

	if (parameters.weightDenseDepth > 0.0f) {
		FindDenseCorrespondences_Kernel << <grid, block >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
		//!!!DEBUGGING
		int* denseCorrCounts = new int[input.maxNumDenseImPairs];
		cutilSafeCall(hipMemcpy(denseCorrCounts, state.d_denseCorrCounts, sizeof(int)*input.maxNumDenseImPairs, hipMemcpyDeviceToHost));
		unsigned int totalCount = 0;
		for (unsigned int i = 0; i < input.maxNumDenseImPairs; i++) totalCount += denseCorrCounts[i];
		if (denseCorrCounts) delete[] denseCorrCounts;
		//!!!DEBUGGING
		BuildDenseDepthSystem_Kernel << <grid, block >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif

		//!!!debugging
		bool debugPrint = false;
		float* h_JtJ = NULL;
		float* h_Jtr = NULL;
		if (debugPrint) {
			h_JtJ = new float[sizeJtJ];
			h_Jtr = new float[sizeJtr];
			cutilSafeCall(hipMemcpy(h_JtJ, state.d_depthJtJ, sizeof(float) * sizeJtJ, hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpy(h_Jtr, state.d_depthJtr, sizeof(float) * sizeJtr, hipMemcpyDeviceToHost));
			printf("JtJ:\n");
			for (unsigned int i = 0; i < 6 * N; i++) {
				for (unsigned int j = 0; j < 6 * N; j++)
					printf(" %f,", h_JtJ[j * 6 * N + i]);
				printf("\n");
			}
			printf("Jtr:\n");
			for (unsigned int i = 0; i < 6 * N; i++) {
				printf(" %f,", h_Jtr[i]);
			}
			printf("\n");
		}
		float sumResidual; int corrCount;
		cutilSafeCall(hipMemcpy(&sumResidual, state.d_sumResidual, sizeof(float), hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpy(&corrCount, state.d_corrCount, sizeof(int), hipMemcpyDeviceToHost));
		printf("\tweight * dense res = %f * %f = %f\t[#corr = %d]\n\n", parameters.weightDenseDepth, sumResidual/parameters.weightDenseDepth, sumResidual, corrCount);
		if (corrCount != totalCount) printf("ERROR: dense image pair corr counts (%d) != total corr count\n", totalCount, corrCount);

		const unsigned int flipgrid = (sizeJtJ + THREADS_PER_BLOCK_DENSE_DEPTH_FLIP - 1) / THREADS_PER_BLOCK_DENSE_DEPTH_FLIP;
		FlipJtJ_Kernel << <flipgrid, THREADS_PER_BLOCK_DENSE_DEPTH_FLIP >> >(sizeJtJ, sizeJtr, state.d_depthJtJ);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif	
		if (debugPrint) {
			cutilSafeCall(hipMemcpy(h_JtJ, state.d_depthJtJ, sizeof(float) * sizeJtJ, hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpy(h_Jtr, state.d_depthJtr, sizeof(float) * sizeJtr, hipMemcpyDeviceToHost));
			printf("JtJ:\n");
			for (unsigned int i = 0; i < 6 * N; i++) {
				for (unsigned int j = 0; j < 6 * N; j++)
					printf(" %f,", h_JtJ[j * 6 * N + i]);
				printf("\n");
			}
			printf("Jtr:\n");
			for (unsigned int i = 0; i < 6 * N; i++) {
				printf(" %f,", h_Jtr[i]);
			}
			printf("\n\n");
			if (h_JtJ) delete[] h_JtJ;
			if (h_Jtr) delete[] h_Jtr;
		}
		//!!!debugging
	}
	if (timer) timer->endEvent();
}

/////////////////////////////////////////////////////////////////////////
// Eval Max Residual
/////////////////////////////////////////////////////////////////////////

__global__ void EvalMaxResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	__shared__ int maxResIndex[THREADS_PER_BLOCK];
	__shared__ float maxRes[THREADS_PER_BLOCK];

	const unsigned int N = input.numberOfCorrespondences * 3; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	maxResIndex[threadIdx.x] = 0;
	maxRes[threadIdx.x] = 0.0f;

	if (x < N) {
		const unsigned int corrIdx = x / 3;
		const unsigned int componentIdx = x - corrIdx * 3;
		float residual = evalResidualDeviceFloat3(corrIdx, componentIdx, input, state, parameters);

		maxRes[threadIdx.x] = residual;
		maxResIndex[threadIdx.x] = x;

		__syncthreads();

		for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {

			if (threadIdx.x < stride) {
				int first = threadIdx.x;
				int second = threadIdx.x + stride;
				if (maxRes[first] < maxRes[second]) {
					maxRes[first] = maxRes[second];
					maxResIndex[first] = maxResIndex[second];
				}
			}

			__syncthreads();
		}

		if (threadIdx.x == 0) {
			//printf("d_maxResidual[%d] = %f (index %d)\n", blockIdx.x, maxRes[0], maxResIndex[0]);
			state.d_maxResidual[blockIdx.x] = maxRes[0];
			state.d_maxResidualIndex[blockIdx.x] = maxResIndex[0];
		}
	}
}

extern "C" void evalMaxResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences * 3; // Number of correspondences (*3 per xyz)
	EvalMaxResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();
}

/////////////////////////////////////////////////////////////////////////
// Eval Cost
/////////////////////////////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float residual = 0.0f;
	if (x < N) {
		residual = evalFDevice(x, input, state, parameters);
		//float out = warpReduce(residual);
		//unsigned int laneid;
		////This command gets the lane ID within the current warp
		//asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
		//if (laneid == 0) {
		//	atomicAdd(&state.d_sumResidual[0], out);
		//}
		atomicAdd(&state.d_sumResidual[0], residual);
	}
}

float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	float residual = 0.0f;

	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	residual = state.getSumResidual();

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
	if (timer) timer->endEvent();

	return residual;
}

/////////////////////////////////////////////////////////////////////////
// Eval Linear Residual
/////////////////////////////////////////////////////////////////////////

//__global__ void SumLinearResDevice(SolverInput input, SolverState state, SolverParameters parameters)
//{
//	const unsigned int N = input.numberOfImages; // Number of block variables
//	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//
//	float residual = 0.0f;
//	if (x > 0 && x < N) {
//		residual = dot(state.d_rRot[x], state.d_rRot[x]) + dot(state.d_rTrans[x], state.d_rTrans[x]);
//		atomicAdd(state.d_sumLinResidual, residual);
//	}
//}
//float EvalLinearRes(SolverInput& input, SolverState& state, SolverParameters& parameters)
//{
//	float residual = 0.0f;
//
//	const unsigned int N = input.numberOfImages;	// Number of block variables
//
//	// Do PCG step
//	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
//
//	float init = 0.0f;
//	cutilSafeCall(hipMemcpy(state.d_sumLinResidual, &init, sizeof(float), hipMemcpyHostToDevice));
//
//	SumLinearResDevice << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
//#ifdef _DEBUG
//	cutilSafeCall(hipDeviceSynchronize());
//	cutilCheckMsg(__FUNCTION__);
//#endif
//
//	cutilSafeCall(hipMemcpy(&residual, state.d_sumLinResidual, sizeof(float), hipMemcpyDeviceToHost));
//	return residual;
//}

/////////////////////////////////////////////////////////////////////////
// Count High Residuals
/////////////////////////////////////////////////////////////////////////

__global__ void CountHighResidualsDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences * 3; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		const unsigned int corrIdx = x / 3;
		const unsigned int componentIdx = x - corrIdx * 3;
		float residual = evalResidualDeviceFloat3(corrIdx, componentIdx, input, state, parameters);

		if (residual > parameters.verifyOptDistThresh)
			atomicAdd(state.d_countHighResidual, 1);
	}
}

extern "C" int countHighResiduals(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences * 3; // Number of correspondences (*3 per xyz)
	cutilSafeCall(hipMemset(state.d_countHighResidual, 0, sizeof(int)));
	CountHighResidualsDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	int count;
	cutilSafeCall(hipMemcpy(&count, state.d_countHighResidual, sizeof(int), hipMemcpyDeviceToHost));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();
	return count;
}




// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	//!!!DEBUGGING
	if (x == 0) {
		state.d_rRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_rTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_pRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_pTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
	}
	else if (x > 0 && x < N)
	//!!!DEBUGGING
	//if (x > 0 && x < N)
	{
		float3 resRot, resTrans;
		evalMinusJTFDevice(x, input, state, parameters, resRot, resTrans);  // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 

		state.d_rRot[x] = resRot;											// store for next iteration
		state.d_rTrans[x] = resTrans;										// store for next iteration

		const float3 pRot = state.d_precondionerRot[x] * resRot;			// apply preconditioner M^-1
		state.d_pRot[x] = pRot;

		const float3 pTrans = state.d_precondionerTrans[x] * resTrans;		// apply preconditioner M^-1
		state.d_pTrans[x] = pTrans;

		d = dot(resRot, pRot) + dot(resTrans, pTrans);						// x-th term of nomimator for computing alpha and denominator for computing beta

		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	//TODO uncomment
	//if (x > 0 && x < N) state.d_rDotzOld[x] = state.d_scanAlpha[0];				// store result for next kernel call

	//!!!DEBUGGING
	if (x == 0) {
		state.d_rDotzOld[x] = 0.0f;
	}
	else if (x > 0 && x < N) {
		state.d_rDotzOld[x] = state.d_scanAlpha[0];
	}
	//!!!DEBUGGING
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	const unsigned int N = input.numberOfImages;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

	if (timer) timer->startEvent("Init1");

	//!!!DEBUGGING //remember to uncomment the delete...
	//float3* rRot = new float3[input.numberOfImages]; // -jtf
	//float3* rTrans = new float3[input.numberOfImages];
	//!!!DEBUGGING

	cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float)));
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif		

	PCGInit_Kernel1 << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif		
	if (timer) timer->endEvent();

	//cutilSafeCall(hipMemcpy(rRot, state.d_rRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(rTrans, state.d_rTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//// print jtr
	//printf("Jtr:\n");
	//for (unsigned int i = 0; i < input.numberOfImages; i++) {
	//	const float3& rotPart = rRot[i]; const float3& transPart = rTrans[i];
	//	printf(" %f, %f, %f, %f, %f, %f,", rotPart.x, rotPart.y, rotPart.z, transPart.x, transPart.y, transPart.z);
	//}
	//printf("\n\n");
	//cutilSafeCall(hipMemcpy(rRot, state.d_pRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(rTrans, state.d_pTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));

	if (timer) timer->startEvent("Init2");
	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(N, state);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();

	//if (rRot) delete[] rRot;
	//if (rTrans) delete[] rTrans;
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

//TODO MAKE EFFICIENT
__global__ void PCGStep_Kernel_Dense(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;

	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTJDevice(x, state, state.d_depthJtJ, input.numberOfImages, rot, trans); // A x p_k  => J^T x J x p_k 

		state.d_Ap_XRot[x] += rot;
		state.d_Ap_XTrans[x] += trans;
	}
}
//__global__ void PCGStep_Kernel_Dense(SolverInput input, SolverState state, SolverParameters parameters)
//{
//	const unsigned int N = input.numberOfImages;							// Number of block variables
//	const unsigned int x = blockIdx.x;
//
//	//float d = 0.0f;
//	if (x > 0 && x < N)
//	{
//		const unsigned int lane = threadIdx.x % WARP_SIZE;
//
//		float3 rot, trans;
//		applyJTJDenseDevice(x, input, state, parameters, rot, trans, threadIdx.x);			// A x p_k  => J^T x J x p_k 
//
//		if (lane == 0)
//		{
//			atomicAdd(&state.d_Ap_XRot[x].x, rot.x);
//			atomicAdd(&state.d_Ap_XRot[x].y, rot.y);
//			atomicAdd(&state.d_Ap_XRot[x].z, rot.z);
//
//			atomicAdd(&state.d_Ap_XTrans[x].x, trans.x);
//			atomicAdd(&state.d_Ap_XTrans[x].y, trans.y);
//			atomicAdd(&state.d_Ap_XTrans[x].z, trans.z);
//		}
//	}
//}

__global__ void PCGStep_Kernel0(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences;					// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		const float3 tmp = applyJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 
		state.d_Jp[x] = tmp;												// store for next kernel call
	}
}

__global__ void PCGStep_Kernel1a(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;
	const unsigned int lane = threadIdx.x % WARP_SIZE;

	//float d = 0.0f;
	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTDevice(x, input, state, parameters, rot, trans, threadIdx.x, lane);			// A x p_k  => J^T x J x p_k 

		if (lane == 0)
		{
			atomicAdd(&state.d_Ap_XRot[x].x, rot.x);
			atomicAdd(&state.d_Ap_XRot[x].y, rot.y);
			atomicAdd(&state.d_Ap_XRot[x].z, rot.z);

			atomicAdd(&state.d_Ap_XTrans[x].x, trans.x);
			atomicAdd(&state.d_Ap_XTrans[x].y, trans.y);
			atomicAdd(&state.d_Ap_XTrans[x].z, trans.z);
		}
	}
}

__global__ void PCGStep_Kernel1b(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;								// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x > 0 && x < N)
	{
		d = dot(state.d_pRot[x], state.d_Ap_XRot[x]) + dot(state.d_pTrans[x], state.d_Ap_XTrans[x]);		// x-th term of denominator of alpha
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	const float dotProduct = state.d_scanAlpha[0];

	float b = 0.0f;
	if (x > 0 && x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;		// update step size alpha

		state.d_deltaRot[x] = state.d_deltaRot[x] + alpha*state.d_pRot[x];			// do a decent step
		state.d_deltaTrans[x] = state.d_deltaTrans[x] + alpha*state.d_pTrans[x];	// do a decent step

		float3 rRot = state.d_rRot[x] - alpha*state.d_Ap_XRot[x];					// update residuum
		state.d_rRot[x] = rRot;														// store for next kernel call

		float3 rTrans = state.d_rTrans[x] - alpha*state.d_Ap_XTrans[x];				// update residuum
		state.d_rTrans[x] = rTrans;													// store for next kernel call

		float3 zRot = state.d_precondionerRot[x] * rRot;							// apply preconditioner M^-1
		state.d_zRot[x] = zRot;														// save for next kernel call

		float3 zTrans = state.d_precondionerTrans[x] * rTrans;						// apply preconditioner M^-1
		state.d_zTrans[x] = zTrans;													// save for next kernel call

		b = dot(zRot, rRot) + dot(zTrans, rTrans);									// compute x-th term of the nominator of beta
	}
	b = warpReduce(b);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(&state.d_scanAlpha[1], b);
	}
}

template<bool lastIteration>
__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N)
	{
		const float rDotzNew = state.d_scanAlpha[1];								// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;				// update step size beta

		state.d_rDotzOld[x] = rDotzNew;											// save new rDotz for next iteration
		state.d_pRot[x] = state.d_zRot[x] + beta*state.d_pRot[x];		// update decent direction
		state.d_pTrans[x] = state.d_zTrans[x] + beta*state.d_pTrans[x];		// update decent direction


		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);

		if (lastIteration)
		{
			state.d_xRot[x] = state.d_xRot[x] + state.d_deltaRot[x];
			state.d_xTrans[x] = state.d_xTrans[x] + state.d_deltaTrans[x];
		}
	}
}

void PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, bool lastIteration, CUDATimer *timer)
{
	const unsigned int N = input.numberOfImages;	// Number of block variables

	// Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

	cutilSafeCall(hipMemset(state.d_scanAlpha, 0, sizeof(float) * 2));

	// sparse part
	if (parameters.weightSparse > 0.0f) {
		const unsigned int Ncorr = input.numberOfCorrespondences; 
		const int blocksPerGridCorr = (Ncorr + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		PCGStep_Kernel0 << <blocksPerGridCorr, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
		PCGStep_Kernel1a << < N, THREADS_PER_BLOCK_JT >> >(input, state, parameters);
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
	}
	if (parameters.weightDenseDepth > 0.0f) {
		//PCGStep_Kernel_Dense << < N, THREADS_PER_BLOCK_JT >> >(input, state, parameters);
		PCGStep_Kernel_Dense << < N, 1 >> >(input, state, parameters); //TODO fix this part
#ifdef _DEBUG
		cutilSafeCall(hipDeviceSynchronize());
		cutilCheckMsg(__FUNCTION__);
#endif
	}
	//float3* Ap_Rot = new float3[input.numberOfImages];
	//float3* Ap_Trans = new float3[input.numberOfImages];
	//cutilSafeCall(hipMemcpy(Ap_Rot, state.d_Ap_XRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(Ap_Trans, state.d_Ap_XTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
	//if (Ap_Rot) delete[] Ap_Rot;
	//if (Ap_Trans) delete[] Ap_Trans;


	PCGStep_Kernel1b << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif


	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif


	if (lastIteration) {
		PCGStep_Kernel3<true> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}
	else {
		PCGStep_Kernel3<false> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}

/////////////////////////////////////////////////////////////////////////
// Apply Update
/////////////////////////////////////////////////////////////////////////

__global__ void ApplyLinearUpdateDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N) {
		state.d_xRot[x] = state.d_xRot[x] + state.d_deltaRot[x];
		state.d_xTrans[x] = state.d_xTrans[x] + state.d_deltaTrans[x];
	}
}

void ApplyLinearUpdate(SolverInput& input, SolverState& state, SolverParameters& parameters)
{
	const unsigned int N = input.numberOfImages; // Number of block variables
	ApplyLinearUpdateDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

extern "C" void solveBundlingStub(SolverInput& input, SolverState& state, SolverParameters& parameters, float* convergenceAnalysis, CUDATimer *timer)
{
	if (convergenceAnalysis) {
		float initialResidual = EvalResidual(input, state, parameters, timer);
		//printf("initial = %f\n", initialResidual);
		convergenceAnalysis[0] = initialResidual; // initial residual
	}
	//unsigned int idx = 0;

	//!!!DEBUGGING
	if (parameters.weightSparse > 0) {
		float initialResidual = EvalResidual(input, state, parameters, timer);
		printf("initial sparse = %f*%f = %f\n", parameters.weightSparse, initialResidual/parameters.weightSparse, initialResidual);
	}
	//!!!DEBUGGING

	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		parameters.weightDenseDepth = parameters.weightDenseDepthInit + nIter * parameters.weightDenseDepthLinFactor;
		BuildDenseDepthSystem(input, state, parameters, timer);
		Initialization(input, state, parameters, timer);

		//float linearResidual = EvalLinearRes(input, state, parameters);
		//linConvergenceAnalysis[idx++] = linearResidual;

		//cutilSafeCall(hipMemcpy(xRot, state.d_pRot, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));
		//cutilSafeCall(hipMemcpy(xTrans, state.d_pTrans, sizeof(float3)*input.numberOfImages, hipMemcpyDeviceToHost));

		for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++)
		{
			PCGIteration(input, state, parameters, linIter == parameters.nLinIterations - 1, timer);

			//linearResidual = EvalLinearRes(input, state, parameters);
			//linConvergenceAnalysis[idx++] = linearResidual;
		}

		//ApplyLinearUpdate(input, state, parameters);	//this should be also done in the last PCGIteration

		//!!!DEBUGGING
		if (parameters.weightSparse > 0) {
			float residual = EvalResidual(input, state, parameters, timer);
			printf("[niter %d] weight * sparse = %f*%f = %f\t[#corr = %d]\n", nIter, parameters.weightSparse, residual/parameters.weightSparse, residual, input.numberOfCorrespondences);
		}
		//!!!DEBUGGING

		if (convergenceAnalysis) {
			float residual = EvalResidual(input, state, parameters, timer);
			convergenceAnalysis[nIter + 1] = residual;
			//printf("[niter %d] %f\n", nIter, residual);
		}
	}
}

////////////////////////////////////////////////////////////////////
// build variables to correspondences lookup
////////////////////////////////////////////////////////////////////

__global__ void BuildVariablesToCorrespondencesTableDevice(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow)
{
	const unsigned int N = numberOfCorrespondences; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		EntryJ& corr = d_correspondences[x];
		if (corr.isValid()) {
			int offset0 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_i], 1); // may overflow - need to check when read
			int offset1 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_j], 1); // may overflow - need to check when read
			if (offset0 < maxNumCorrespondencesPerImage && offset1 < maxNumCorrespondencesPerImage)	{
				d_variablesToCorrespondences[corr.imgIdx_i * maxNumCorrespondencesPerImage + offset0] = x;
				d_variablesToCorrespondences[corr.imgIdx_j * maxNumCorrespondencesPerImage + offset1] = x;
			}
			else { //invalidate
				corr.setInvalid(); //make sure j corresponds to jt
			}
		}
	}
}

extern "C" void buildVariablesToCorrespondencesTableCUDA(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow, CUDATimer* timer)
{
	const unsigned int N = numberOfCorrespondences;

	if (timer) timer->startEvent(__FUNCTION__);

	BuildVariablesToCorrespondencesTableDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_correspondences, numberOfCorrespondences, maxNumCorrespondencesPerImage, d_variablesToCorrespondences, d_numEntriesPerRow);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

	if (timer) timer->endEvent();
}
